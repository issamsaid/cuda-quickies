
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// these are just for timing measurments
#include <time.h>
// Computes minimum in a 3D volume, at each output point
// To compile it with nvcc execute: nvcc -O2 -o grid3d grid3d.cu
//define the window size (cubic volume) and the data set size
#define WSIZE 6
#define DATAXSIZE 100
#define DATAYSIZE 100
#define DATAZSIZE 20
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)
// device function to compute 3D volume minimum at each output point
__global__ void cmp_win(int knode[][DATAYSIZE][DATAXSIZE], const int kcell[][DATAYSIZE+(WSIZE-1)][DATAXSIZE+(WSIZE-1)])
{
    __shared__ int smem[(BLKZSIZE + (WSIZE-1))][(BLKYSIZE + (WSIZE-1))][(BLKXSIZE + (WSIZE-1))];
    int tempnode, i, j, k;
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if ((idx < (DATAXSIZE+WSIZE-1)) && (idy < (DATAYSIZE+WSIZE-1)) && (idz < (DATAZSIZE+WSIZE-1))){
      smem[threadIdx.z][threadIdx.y][threadIdx.x]=kcell[idz][idy][idx];
      if ((threadIdx.z > (BLKZSIZE - WSIZE)) && (idz < DATAZSIZE))
        smem[threadIdx.z + (WSIZE-1)][threadIdx.y][threadIdx.x] = kcell[idz + (WSIZE-1)][idy][idx];
      if ((threadIdx.y > (BLKYSIZE - WSIZE)) && (idy < DATAYSIZE))
        smem[threadIdx.z][threadIdx.y + (WSIZE-1)][threadIdx.x] = kcell[idz][idy+(WSIZE-1)][idx];
      if ((threadIdx.x > (BLKXSIZE - WSIZE)) && (idx < DATAXSIZE))
        smem[threadIdx.z][threadIdx.y][threadIdx.x + (WSIZE-1)] = kcell[idz][idy][idx+(WSIZE-1)];
      if ((threadIdx.z > (BLKZSIZE - WSIZE)) && (threadIdx.y > (BLKYSIZE - WSIZE)) && (idz < DATAZSIZE) && (idy < DATAYSIZE))
        smem[threadIdx.z + (WSIZE-1)][threadIdx.y + (WSIZE-1)][threadIdx.x] = kcell[idz+(WSIZE-1)][idy+(WSIZE-1)][idx];
      if ((threadIdx.z > (BLKZSIZE - WSIZE)) && (threadIdx.x > (BLKXSIZE - WSIZE)) && (idz < DATAZSIZE) && (idx < DATAXSIZE))
        smem[threadIdx.z + (WSIZE-1)][threadIdx.y][threadIdx.x + (WSIZE-1)] = kcell[idz+(WSIZE-1)][idy][idx+(WSIZE-1)];
      if ((threadIdx.y > (BLKYSIZE - WSIZE)) && (threadIdx.x > (BLKXSIZE - WSIZE)) && (idy < DATAYSIZE) && (idx < DATAXSIZE))
        smem[threadIdx.z][threadIdx.y + (WSIZE-1)][threadIdx.x + (WSIZE-1)] = kcell[idz][idy+(WSIZE-1)][idx+(WSIZE-1)];
      if ((threadIdx.z > (BLKZSIZE - WSIZE)) && (threadIdx.y > (BLKYSIZE - WSIZE)) && (threadIdx.x > (BLKXSIZE - WSIZE)) && (idz < DATAZSIZE) && (idy < DATAYSIZE) && (idx < DATAXSIZE))
        smem[threadIdx.z+(WSIZE-1)][threadIdx.y+(WSIZE-1)][threadIdx.x+(WSIZE-1)] = kcell[idz+(WSIZE-1)][idy+(WSIZE-1)][idx+(WSIZE-1)];
      }
    __syncthreads();
    if ((idx < DATAXSIZE) && (idy < DATAYSIZE) && (idz < DATAZSIZE)){
      tempnode = knode[idz][idy][idx];
      for (i=0; i<WSIZE; i++)
        for (j=0; j<WSIZE; j++)
          for (k=0; k<WSIZE; k++)
          if (smem[threadIdx.z + i][threadIdx.y + j][threadIdx.x + k] < tempnode)
            tempnode = smem[threadIdx.z + i][threadIdx.y + j][threadIdx.x + k];
      knode[idz][idy][idx] = tempnode;
      }
}

int main(int argc, char *argv[])
{
    typedef int cRarray[DATAYSIZE+WSIZE-1][DATAXSIZE+WSIZE-1];
    typedef int nRarray[DATAYSIZE][DATAXSIZE];
    int i, j, k, u, v, w, temphnode;
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize(((DATAXSIZE+BLKXSIZE-1)/BLKXSIZE), ((DATAYSIZE+BLKYSIZE-1)/BLKYSIZE), ((DATAZSIZE+BLKZSIZE-1)/BLKZSIZE));
// these are just for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0f;
    double t2sum=0.0f;
    double t3sum=0.0f;
// overall data set sizes
    const int nx = DATAXSIZE;
    const int ny = DATAYSIZE;
    const int nz = DATAZSIZE;
// window (cubic minimization volume) dimensions
    const int wx = WSIZE;
    const int wy = WSIZE;
    const int wz = WSIZE;
// pointers for data set storage via malloc
    nRarray *hnode; // storage for result computed on host
    nRarray *node, *d_node;  // storage for result computed on device
    cRarray *cell, *d_cell;  // storage for input
// start timing
    t0 = clock();
// allocate storage for data set
    if ((cell = (cRarray *)malloc(((nx+(wx-1))*(ny+(wy-1))*(nz+(wz-1)))*sizeof(int))) == 0) {fprintf(stderr,"malloc Fail \n"); return 1;}
    if ((node = (nRarray *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc Fail \n"); return 1; }
    if ((hnode = (nRarray *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr, "malloc Fail \n"); return 1; }
// synthesize data
    for(i=0; i<(nz+(wz-1)); i++)
      for(j=0; j<(ny+(wy-1)); j++)
        for (k=0; k<(nx+(wx-1)); k++){
          cell[i][j][k] = rand(); // unless we use a seed this will produce the same sequence all the time
          if ((i<nz) && (j<ny) && (k<nx)) {
            node[i][j][k]  = RAND_MAX;
            hnode[i][j][k] = RAND_MAX;
            }
          }
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %3.2f seconds.  Begin compute\n", t1sum);
// allocate GPU device buffers
    hipMalloc((void **) &d_cell, (((nx+(wx-1))*(ny+(wy-1))*(nz+(wz-1)))*sizeof(int)));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_node, ((nx*ny*nz)*sizeof(int)));
    cudaCheckErrors("Failed to allocate device buffer2");
// copy data to GPU
    hipMemcpy(d_node, node, ((nx*ny*nz)*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");
    hipMemcpy(d_cell, cell, (((nx+(wx-1))*(ny+(wy-1))*(nz+(wz-1)))*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy2 failure");

    cmp_win<<<gridSize,blockSize>>>(d_node, d_cell);
    cudaCheckErrors("Kernel launch failure");
// copy output data back to host

    hipMemcpy(node, d_node, ((nx*ny*nz)*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy3 failure");
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf(" Device compute took %3.2f seconds.  Beginning host compute.\n", t2sum);
// now compute the same result on the host
    for (u=0; u<nz; u++)
      for (v=0; v<ny; v++)
        for (w=0; w<nx; w++){
          temphnode = hnode[u][v][w];
          for (i=0; i<wz; i++)
            for (j=0; j<wy; j++)
              for (k=0; k<wx; k++)
                if (temphnode > cell[i+u][j+v][k+w]) temphnode = cell[i+u][j+v][k+w];
          hnode[u][v][w] = temphnode;
          }
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf(" Host compute took %3.2f seconds.  Comparing results.\n", t3sum);
// and compare for accuracy
    for (i=0; i<nz; i++)
      for (j=0; j<ny; j++)
        for (k=0; k<nx; k++)
          if (hnode[i][j][k] != node[i][j][k]) {
            printf("Mismatch at x= %d, y= %d, z= %d  Host= %d, Device = %d\n", i, j, k, hnode[i][j][k], node[i][j][k]);
            return 1;
            }
    printf("Results match!\n");
    free(cell);
    free(node);
    hipFree(d_cell);
    cudaCheckErrors("hipFree fail");
    hipFree(d_node);
    cudaCheckErrors("hipFree fail");
    return 0;
}
